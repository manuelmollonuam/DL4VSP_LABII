#include "hip/hip_runtime.h"
#include "../include/kernel.h"

template<typename T> struct TtoInt { static const int test = -1; };
template<> struct TtoInt<float> { static const int test = 0; }; 
template<> struct TtoInt<half> { static const int test = 0; }; 
template<> struct TtoInt<double> { static const int test = 0; }; 

#if __CUDACC_VER_MAJOR__ >= 9
#define __SHFL_DOWN(var, delta)  __shfl_down_sync(0xffffffff, var, delta)
#else
#define __SHFL_DOWN(var, delta)  __shfl_down(var, delta)
#endif

#if __CUDACC_VER_MAJOR__ >= 9
#define __SYNCWARP __syncwarp()
#else
#define __SYNCWARP 
#endif

// Block size for weight_norm_*_first_dim_kernel.
// Currently, kernels are non-persistent.
// Dialing up the block size to, say 1024, can improve performance by
// increase the amount of cache available per block, which can improve cache hit rate.
// However, this is less efficient for short rows.  256 is pretty versatile. 
// May be worth implementing heuristics later.
#define BLOCK 256

// Block size for weight_norm_*_last_dim_kernel.
// This is tricker than the first_dim case because we must make blocks 
// at least 16 fast elements wide to ensure fully-coalesced half-precision accesses.
// Since output-element parallelism is along the fast dimension, this reduces the number of 
// blocks we can launch by 16X.  
#define TILE_W 16
// Somewhat versatile strategy: max out intra-block parallelism by extending
// blocks across the slow dimension up to the hardware-max block size of 1024.
#define TILE_H 64

using namespace std;

// lanes is intended to be <= 32.
template <typename T>
__device__ __forceinline__ void reduce_block_into_lanes(T *x, T val, int lanes) 
{ 
  int tid = threadIdx.x + threadIdx.y*blockDim.x;
  int blockSize = blockDim.x*blockDim.y;

  if(blockSize >= 64)
  {
    x[tid] = val;
    __syncthreads();
  }
  
  #pragma unroll
  for(int i = (blockSize >> 1); i >= 64; i >>= 1) 
  {
    if(tid < i)
      x[tid] += x[tid+i]; // JoinOp
    __syncthreads();
  }

  if(tid < 32) 
  {
    T final;
    if(blockSize >= 64)
      final = x[tid] + x[tid+32]; // JoinOp
    else
      final = val;
    // __SYNCWARP();

    #pragma unroll
    for(int i = 16; i >= lanes; i >>= 1)
      final += __SHFL_DOWN(final, i);

    if(tid < lanes) 
      x[tid] = final; // EpilogueOp
  }

  // Make sure the smem result is visible to all warps.
  __syncthreads();
}

template <typename T, typename IndexType>
__global__ void weight_norm_fwd_first_dim_kernel
(
  TensorInfo<T, IndexType> w,
  TensorInfo<float, IndexType> norms,
  TensorInfo<T, IndexType> v,
  TensorInfo<T, IndexType> g,
  IndexType rowSize
)
{
  // We are norming each slowest-dim row of the tensor separately.
  // For now, assign one block to each row.
  IndexType tid = threadIdx.x;
  IndexType row = blockIdx.x;
  IndexType stride = blockDim.x;

  // Logical index offset for this flattened row
  IndexType rowStart = row*rowSize;

  extern __shared__ float s[];
  
  float thread_sum = 0.f;
  for(IndexType i = tid; i < rowSize; i += stride ) 
  {
    float val_f = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(v, i + rowStart)); 
    thread_sum += val_f*val_f; // AccumOp, could do Kahan here
  }

  reduce_block_into_lanes(s, thread_sum, 1);
  float result = s[0];

  result = sqrtf(result);
  
  if(tid == 0)
    DEVICE_LINEAR_GET_F(norms, row) = result;

  // Broadcast load, could use shared memory instead.
  float g_this_row = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(g, row));

  float rnorm = 1.f/result; // for consistency with backward kernel

  // Write data to output
  for(IndexType i = tid; i < rowSize; i += stride ) 
  {
    float val_f = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(v, i + rowStart));
    DEVICE_LINEAR_GET(w, i + rowStart) = ScalarConvert<float,T>::to(g_this_row*val_f*rnorm);
  }
}

template <typename T, typename IndexType>
__global__ void weight_norm_fwd_last_dim_kernel
(
  TensorInfo<T, IndexType> w,
  TensorInfo<float, IndexType> norms,
  TensorInfo<T, IndexType> v,
  TensorInfo<T, IndexType> g,
  IndexType fast_dim_size,
  IndexType slower_dims_size
)
{
  IndexType fast_dim_location = threadIdx.x + blockIdx.x*blockDim.x;

  extern __shared__ float alloc[];
  float* s = &alloc[0];
  float* rnorms_this_block = &alloc[blockDim.x*blockDim.y];

  float thread_sum = 0.f;

  IndexType slower_dims_location = threadIdx.y;
  IndexType currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      float val_f = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(v, currentIdx)); 
      thread_sum += val_f*val_f; // AccumOp, could do Kahan here
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y; 
    }

  reduce_block_into_lanes(s, thread_sum, blockDim.x); 

  // Better to pass an EpilogueOp to reduce_block_into_lanes, implement later
  if(threadIdx.y == 0)
  {
    float result = s[threadIdx.x];
    float norm_this_col = sqrtf(result);
    DEVICE_LINEAR_GET_F(norms, fast_dim_location) = norm_this_col;
    rnorms_this_block[threadIdx.x] = 1.f/norm_this_col;
  }
   
  __syncthreads(); 

  float g_this_col = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(g, fast_dim_location));     

  float rnorm = rnorms_this_block[threadIdx.x]; 

  slower_dims_location = threadIdx.y;
  currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      float val_f = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(v, currentIdx)); 
      DEVICE_LINEAR_GET(w, currentIdx) = ScalarConvert<float,T>::to(g_this_col*val_f*rnorm);
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y; 
    } 
}

template <typename T, typename IndexType>
__global__ void weight_norm_bwd_first_dim_kernel
(
  TensorInfo<T, IndexType> pLpv,
  TensorInfo<T, IndexType> pLpg,
  TensorInfo<T, IndexType> pLpw,
  TensorInfo<T, IndexType> savedv,
  TensorInfo<T, IndexType> savedg,
  TensorInfo<float, IndexType> savedNorms,
  IndexType rowSize
)
{
  // For now, assign one block to each row.
  IndexType tid = threadIdx.x;
  IndexType row = blockIdx.x;
  IndexType stride = blockDim.x;

  // Logical index offset for this flattened row
  IndexType rowStart = row*rowSize;

  extern __shared__ float s[];
  
  float thread_sum = 0.f;
  for(IndexType i = tid; i < rowSize; i += stride ) 
  {
    float pLpwi = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(pLpw, i + rowStart)); 
    float savedvi = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedv, i + rowStart)); 
    thread_sum += pLpwi*savedvi; // AccumOp, could do Kahan here
  }

  reduce_block_into_lanes(s, thread_sum, 1);
  float result = s[0];

  // Could choose to save reciprocal of norm instead I suppose, but norms is probably
  // more handy to keep around.
  // Broadcast load; could use shared memory instead.
  float rnorm = 1.f/DEVICE_LINEAR_GET_F(savedNorms, row);  
  float rnorm3 = rnorm*rnorm*rnorm;

  // Write g gradients.
  if(tid == 0)
    DEVICE_LINEAR_GET(pLpg, row) = ScalarConvert<float, T>::to(result*rnorm);

  // Broadcast load, could use shared memory instead.
  float g_this_row = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedg, row));
   
  // Write v gradients.  We are reusing values that were loaded earlier, so there 
  // is an optimization opportunity here (store values persistently).
  for(IndexType j = tid; j < rowSize; j += stride ) 
  {
    float pLpwj = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(pLpw, j + rowStart));  
    float savedvj = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedv, j + rowStart));  
    float pLpvj = g_this_row*(rnorm*pLpwj - rnorm3*savedvj*result);
    DEVICE_LINEAR_GET(pLpv, j + rowStart) = ScalarConvert<float,T>::to(pLpvj);
  }
}

template <typename T, typename IndexType>
__global__ void weight_norm_bwd_last_dim_kernel
(
  TensorInfo<T, IndexType> pLpv,
  TensorInfo<T, IndexType> pLpg,
  TensorInfo<T, IndexType> pLpw,
  TensorInfo<T, IndexType> savedv,
  TensorInfo<T, IndexType> savedg,
  TensorInfo<float, IndexType> savedNorms,
  IndexType fast_dim_size,
  IndexType slower_dims_size
)
{
  IndexType fast_dim_location = threadIdx.x + blockIdx.x*blockDim.x;

  extern __shared__ float s[];

  float thread_sum = 0.f;

  IndexType slower_dims_location = threadIdx.y;
  IndexType currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      float pLpwi = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(pLpw, currentIdx)); 
      float savedvi = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedv, currentIdx)); 
      thread_sum += pLpwi*savedvi; // AccumOp, could do Kahan here
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y; 
    }

  reduce_block_into_lanes(s, thread_sum, blockDim.x); 
  float result = s[threadIdx.x];

  // Broadcast load; could use shared memory instead.
  float rnorm = 1.f/DEVICE_LINEAR_GET_F(savedNorms, fast_dim_location);  
  float rnorm3 = rnorm*rnorm*rnorm;

  // Write g gradients.
  if(threadIdx.y == 0)
    DEVICE_LINEAR_GET(pLpg, fast_dim_location) = ScalarConvert<float, T>::to(result*rnorm);

  // Entire block pulls these values, could use shared memory instead.
  float g_this_col = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedg, fast_dim_location));

  // Write v gradients.
  slower_dims_location = threadIdx.y;
  currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      float pLpwj = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(pLpw, currentIdx));  
      float savedvj = ScalarConvert<T, float>::to(DEVICE_LINEAR_GET(savedv, currentIdx));  
      float pLpvj = g_this_col*(rnorm*pLpwj - rnorm3*savedvj*result);
      DEVICE_LINEAR_GET(pLpv, currentIdx) = ScalarConvert<float,T>::to(pLpvj);
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y; 
    } 
}

template<typename DataType, 
         typename AccumType,
         typename IndexType>
void send_to_fwd_wrapper::call
(
  vector<TensorInfo<void, idxType>>& tensors,
  int dim
)
{
#ifdef DEBUG_ANY
  cout << "hello from send_to_fwd with v.type = " << v.type << endl;
#endif

  auto w    (*((TensorInfo<DataType , idxType>*)&tensors[0]));
  auto norms(*((TensorInfo<AccumType, idxType>*)&tensors[1]));
  auto v    (*((TensorInfo<DataType , idxType>*)&tensors[2]));
  auto g    (*((TensorInfo<DataType , idxType>*)&tensors[3]));

  if(dim == 0) 
  {
    // Find logical size of each flattened slowest-dim row
    IndexType rowSize = 1;
    for(IndexType i = v.dims - 1; i > 0; i--)
      rowSize *= v.sizes[i];

    weight_norm_fwd_first_dim_kernel<<<v.sizes[0], BLOCK, BLOCK*sizeof(float)>>>
    (
      w, 
      norms,
      v,  
      g,  
      rowSize
    );
  }
  else if(dim == v.dims - 1)
  {
    // Precompute slower_dims_size and fast_dim_size because they involve dynamically indexing an array.
    IndexType slower_dims_size = 1;
    for(IndexType i = 0; i < v.dims - 1; i++)
      slower_dims_size *= v.sizes[i];

    int fast_dim_size = v.sizes[v.dims-1];

    weight_norm_fwd_last_dim_kernel<<<(fast_dim_size+TILE_W-1)/TILE_W,
                                       dim3(TILE_W,TILE_H), 
                                       (TILE_W*TILE_H + TILE_W)*sizeof(float)>>>
    (
      w, 
      norms,
      v,  
      g,  
      fast_dim_size,
      slower_dims_size 
    );
  }
  // else 
  // {
  //   intermediate dim kernel.  Error checking on the dim was already done in 
  //   Module.cpp:weight_norm_fwd.  Could put that logic here instead, if we include
  //   <python.h> in both files.
  // }

#ifdef DEBUG_PROFILE
  hipDeviceSynchronize();
#endif
}

template<typename DataType,
         typename AccumType,
         typename IndexType>
void send_to_bwd_wrapper::call
(
  vector<TensorInfo<void, idxType>>& tensors,
  int dim
)
{
#ifdef DEBUG_ANY
  cout << "Hello from send_to_bwd with pLpw.type = " << pLpw.type << endl;
#endif

  auto pLpv      (*((TensorInfo<DataType , idxType>*)&tensors[0]));
  auto pLpg      (*((TensorInfo<DataType , idxType>*)&tensors[1]));
  auto pLpw      (*((TensorInfo<DataType , idxType>*)&tensors[2]));
  auto savedv    (*((TensorInfo<DataType , idxType>*)&tensors[3]));
  auto savedg    (*((TensorInfo<DataType , idxType>*)&tensors[4]));
  auto savedNorms(*((TensorInfo<AccumType, idxType>*)&tensors[5]));

  if(dim == 0) 
  {
    // Find logical size of each flattened slowest-dim row
    IndexType rowSize = 1;
    for(IndexType i = savedv.dims - 1; i > 0; i--)
      rowSize *= savedv.sizes[i];

    weight_norm_bwd_first_dim_kernel<<<pLpw.sizes[0], BLOCK, BLOCK*sizeof(float)>>>
    (
      pLpv,
      pLpg,
      pLpw,
      savedv,
      savedg,
      savedNorms,
      rowSize
    );
  }
  else if(dim == savedv.dims - 1)
  {
    // Precompute slower_dims_size and fast_dim_size because they involve dynamically indexing an array.
    IndexType slower_dims_size = 1;
    for(IndexType i = 0; i < savedv.dims - 1; i++)
      slower_dims_size *= savedv.sizes[i];

    int fast_dim_size = savedv.sizes[savedv.dims-1];

    weight_norm_bwd_last_dim_kernel<<<(fast_dim_size+TILE_W-1)/TILE_W,
                                       dim3(TILE_W,TILE_H), 
                                       (TILE_W*TILE_H + TILE_W)*sizeof(float)>>>
    (
      pLpv,
      pLpg,
      pLpw,
      savedv,
      savedg,
      savedNorms,
      fast_dim_size,
      slower_dims_size 
    );
  }
  // else 
  // {
  //   intermediate dim kernel.  Error checking on the dim was already done in 
  //   Module.cpp:weight_norm_bwd.  Could put that logic here instead, if we include
  //   <python.h> in both files.
  // }

#ifdef DEBUG_PROFILE
  hipDeviceSynchronize();
#endif
}

#define INSTANTIATE_SEND_TO_FWD(DATATYPE, ACCUMTYPE, IDXTYPE)         \
template void send_to_fwd_wrapper::call<DATATYPE, ACCUMTYPE, IDXTYPE> \
(                                                                     \
  vector<TensorInfo<void, idxType>>&,                                 \
  int                                                                 \
);
INSTANTIATE_SEND_TO_FWD(float, float, idxType)
INSTANTIATE_SEND_TO_FWD(half, float, idxType)
#undef INSTANTIATE_SEND_TO_FWD

#define INSTANTIATE_SEND_TO_BWD(DATATYPE, ACCUMTYPE, IDXTYPE)         \
template void send_to_bwd_wrapper::call<DATATYPE, ACCUMTYPE, IDXTYPE> \
(                                                                     \
  vector<TensorInfo<void, idxType>>&,                                 \
  int                                                                 \
);                                                            
INSTANTIATE_SEND_TO_BWD(float, float, idxType)
INSTANTIATE_SEND_TO_BWD(half, float, idxType)
#undef INSTANTIATE_SEND_TO_BWD

#undef BLOCK
#undef TILE_W
#undef TILE_H
